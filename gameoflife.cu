#include "hip/hip_runtime.h"
#include "common/book.h"
#include "common/cpu_bitmap.h"

#include "hip/hip_runtime.h"
#include "cuda_gl_interop.h"

int *field_read_d;
int *field_writ_d;
int *field_rdwr_h;

PFNGLBINDBUFFERARBPROC    glBindBuffer     = NULL;
PFNGLDELETEBUFFERSARBPROC glDeleteBuffers  = NULL;
PFNGLGENBUFFERSARBPROC    glGenBuffers     = NULL;
PFNGLBUFFERDATAARBPROC    glBufferData     = NULL;

GLuint  bufferObj;
hipGraphicsResource *resource;

uchar4 *pixMap;

int ROWS = 1024;
int COLS = 1024;

float zoom = 0.0;
float dx   = 0.0;
float dy   = 0.0;

/* Utilitary fucntion to count neighbourds around a cell.*/
__device__ int
count_alive( int *field, int i, int j, int ROWS, int COLS)
{
   int x, y, a=0;
   for(x=i-1; x <= (i+1) ; x++)
   {
      for(y=j-1; y <= (j+1) ; y++)
      {
         if ( (x==i) && (y==j) ) continue;
         if ( (y<ROWS) && (x<COLS) && (x>=0)   && (y>=0) )
         {
              a += field[ROWS*y+x];
         }
      }
   }
   
   return a;
	
}

/* Evolve step applies the Game Of Life rules to each pixel and the 
 * ones around to check if the cell keeps living or dies.
 */
__global__ void
evolveStep ( int *field_read, int *field_write, int ROWS, int COLS )
{
	
	int  x = threadIdx.x + blockIdx.x*blockDim.x;
	int  y = threadIdx.y + blockIdx.y*blockDim.y;
	
	int neigh = count_alive( field_read, x, y, ROWS, COLS );
	
	if( field_read[y*ROWS + x] ){
		if( (neigh > 3) || (neigh<2) )
			field_write[ y*ROWS + x ] = 0;
		else
			field_write[ y*ROWS + x ] = 1;
	}
	else{
		if( neigh == 3  )
			field_write[ y*ROWS + x ] = 1;
		else
			field_write[ y*ROWS + x ] = 0;
	}
	
}

/* This function transforms the unidimensional array into a bitmap buffer
 * using the GPU parallelism to speed up the process
 */
__global__ void 
toPixelMap ( uchar4 *ptr, int *field ) 
{
   
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

	if( field[offset] ){
		ptr[offset].x = 255;
		ptr[offset].y = 255;
		ptr[offset].z = 255;
		ptr[offset].w = 255;
	}
	else{
		ptr[offset].x = 0;
		ptr[offset].y = 0;
		ptr[offset].z = 0;
		ptr[offset].w = 0;
	}
}

void generate_frame( uchar4 *pixels ) 
{
    dim3    grids(ROWS/16,COLS/16);
    dim3    threads(16,16);
    
    /*Evolve step is a kernel*/
    evolveStep <<< grids, threads >>>( field_read_d, field_writ_d, ROWS, COLS );
    /*toPixelMap is another kernel!*/     
    toPixelMap <<< grids, threads >>>( pixels, field_writ_d                   );
    /* Swap buffer; copy the field_write to field_read.
     * This is neccesary since parallel execution and pixel interdepenceny
     * does not guarantee we can operate in "serial mode" without messing 
     * things up badly!
     */	
    hipMemcpy( field_read_d, field_writ_d, sizeof(int)*ROWS*COLS, hipMemcpyDeviceToDevice );
	
}
	
static void key_func( unsigned char key, int x, int y ) 
{
    /*ESC key terminates program*/
    switch (key) {
        case 27:
            // clean up OpenGL and CUDA
            HANDLE_ERROR( hipGraphicsUnregisterResource( resource ) );
            glBindBuffer( GL_PIXEL_UNPACK_BUFFER_ARB, 0 );
            glDeleteBuffers( 1, &bufferObj );
            exit(0);
    }
    
    /*Just to zoom around*/
    if( key == 'a' )
		dx +=    0.5;
	if( key == 'd' )
		dx +=   -0.5;
	if( key == 'w' )
		dy +=   -0.5;
	if( key == 's' )
		dy +=    0.5;
	if( key == 'i' )
		zoom +=  0.5;
	if( key == 'j' )
		zoom += -0.5;
}

static void draw_func( void ) 
{
   	
    HANDLE_ERROR( hipGraphicsMapResources( 1, &resource, NULL ) );
    
    size_t  size;
    
    HANDLE_ERROR( hipGraphicsResourceGetMappedPointer( (void**)&pixMap, &size, resource) );
	generate_frame( pixMap );
    
    HANDLE_ERROR( hipGraphicsUnmapResources( 1, &resource, NULL ) );
    
    glDrawPixels( ROWS, COLS, GL_RGBA, GL_UNSIGNED_BYTE, 0 );
    
    glutSwapBuffers();
    glutPostRedisplay();
}


int main( int argc, char **argv ) 
{
    hipDeviceProp_t  prop;
    int dev;

    memset( &prop, 0, sizeof( hipDeviceProp_t ) );
    prop.major = 1;
    prop.minor = 0;
    HANDLE_ERROR( hipChooseDevice( &dev, &prop ) );
    HANDLE_ERROR( cudaGLSetGLDevice( dev ) );

    glutInit( &argc, argv );
    glutInitDisplayMode( GLUT_DOUBLE | GLUT_RGBA );
    glutInitWindowSize( ROWS, COLS );
    glutCreateWindow( "bitmap" );

    /*Bitmap buffer bullshit*/
    /*************************************************************************************/
    glBindBuffer    = (PFNGLBINDBUFFERARBPROC)   GET_PROC_ADDRESS("glBindBuffer");
    glDeleteBuffers = (PFNGLDELETEBUFFERSARBPROC)GET_PROC_ADDRESS("glDeleteBuffers");
    glGenBuffers    = (PFNGLGENBUFFERSARBPROC)   GET_PROC_ADDRESS("glGenBuffers");
    glBufferData    = (PFNGLBUFFERDATAARBPROC)   GET_PROC_ADDRESS("glBufferData");
    glGenBuffers( 1, &bufferObj );
    glBindBuffer( GL_PIXEL_UNPACK_BUFFER_ARB, bufferObj );
    glBufferData( GL_PIXEL_UNPACK_BUFFER_ARB, ROWS * COLS * 4, NULL, GL_DYNAMIC_DRAW_ARB );
    HANDLE_ERROR(    hipGraphicsGLRegisterBuffer( &resource , cudaGraphicsMapFlagsNone ) );
    /*************************************************************************************/
    
    /*Array to hold cell status*/
    field_rdwr_h = (int *)malloc( sizeof(int)*ROWS*COLS );
    /*Initialize with 0s and 1s*/
    for( int i=0; i<ROWS*COLS; i++ ) 
      field_rdwr_h[i] = rand()%2;
    
    /* Allocate memory on device: note we have 2 buffers, to read an write!*/
    hipMalloc( (void **)&field_read_d, sizeof(int)*ROWS*COLS );
    hipMalloc( (void **)&field_writ_d, sizeof(int)*ROWS*COLS );
    /*Copy stuff around*/ 
    hipMemcpy( field_read_d, field_rdwr_h, sizeof(int)*ROWS*COLS, hipMemcpyHostToDevice );
    
    /*Attach keyboard function*/
    glutKeyboardFunc( key_func );
    /*Attach display function, yes, a pointer to a function LoL*/
    glutDisplayFunc( draw_func );
    /*Start loop*/
    glutMainLoop();
}
